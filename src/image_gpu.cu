// dummy image 
#include "image_gpu.h"
#include <hip/hip_runtime.h>


// TBD REF
static void* g_imgSrc = nullptr;
static void* g_imgDst = nullptr;
static void* g_imgAux = nullptr;


// TBD REF, use context
bool isCUDA_OK()
{
    int nCnt = 0;
    bool bRet = false;

    hipError_t error_id = hipGetDeviceCount(&nCnt);
    if (error_id == hipSuccess)
    {
        bRet = (nCnt > 0);
    }
    return bRet;
}

bool cudaClose()
{
    if( g_imgSrc)    
    {
        hipFree(g_imgSrc);
    }
    if( g_imgDst)    
    {
        hipFree(g_imgDst);
    }
    if( g_imgAux)    
    {
        hipFree(g_imgAux);
    }
}

bool cudaGrayscale( unsigned char* pImageSrc, int nWidth, int nHeight)
{
    bool bRet = false;
    
    if( !pImageSrc)
    {
        return bRet;
    }

    // alloc buffers
    if( !g_imgSrc)
    {
        hipError_t error_id = hipMalloc( (void **)&g_imgSrc, nWidth * nHeight * 4 ); // assume RGBA
    }
    if( !g_imgSrc)
    {
        return bRet;
    }

    if( !g_imgDst)
    {
        hipError_t error_id = hipMalloc( (void **)&g_imgDst, nWidth * nHeight); // assume GS
    }
    if( !g_imgDst)
    {
        return bRet;
    }

    if( !g_imgAux)
    {
        hipError_t error_id = hipMalloc( (void **)&g_imgAux, nWidth * nHeight); // assume GS
    }
    if( !g_imgAux)
    {
        return bRet;
    }

    // load data
    hipError_t error_id = hipMemcpy( g_imgSrc, pImageSrc, nWidth * nHeight * 4, hipMemcpyHostToDevice );

    // start processing

    bRet = true;

    return bRet;
}

bool cudaBlur( unsigned char* pImageDst, int nWidth, int nHeight)
{
    bool bRet = false;

    if( !pImageDst)
    {
        return bRet;
    }

    if( !g_imgDst || !g_imgAux)
    {
        return bRet;
    }

    // start processing
    
    bRet = true;
    // save data
    hipError_t error_id = hipMemcpy( (void*)pImageDst, g_imgDst, nWidth * nHeight, hipMemcpyDeviceToHost );

    return bRet;
}







